#include "hip/hip_runtime.h"
#include "bvh.h"
#include "aabb.h"
#include "common.h"
#include <thrust/device_ptr.h>
#include <thrust/device_free.h>

#define TEXTURE_GPU (0)
#define QUICK_TRANS (1)

#define HEAT_SOURCE_SIZE (3)
#define HEAT_TRANSFER_SPEED (0.10f)

volatile int dstOut = -1;
float isource = 1.0f;

int heatSource[HEAT_SOURCE_SIZE] = {100, 10, 200};


/* heat and color */
float* gIntensity_h_; // pined memory
float* gIntensityIn_d_;
float* gIntensityOut_d_;

#if TEXTURE_GPU
texture<float>  texIn;
#endif

#if STEAM_GPU
hipStream_t stream1;
#endif

/* global value */
extern std::uint32_t gNumObjects;
extern std::uint32_t* gSortedObjIDs;

lbvh::BVH* bvhInstance = lbvh::BVH::getInstance();
/* print info */
std::string div_signs(10, '-');

#if TEXTURE_GPU
    __global__ void 
    propagate_Kernel(std::uint32_t num_objects, std::uint32_t* adjObjects, std::uint32_t* sortedIDs,
            std::uint32_t* prefix_sum, std::uint32_t* adjObjNums, float* curr) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx > num_objects - 1) {
            return;
        }
        std::uint32_t adjObjNum = adjObjNums[idx];

        float curr_tmp = 0.6 * tex1Dfetch(texIn, idx); // heat loss
        for (int i = 0; i < adjObjNum; i++) {
            curr_tmp +=  tex1Dfetch(texIn, adjObjects[prefix_sum[idx]/*offset*/ + i]);
        }
        curr_tmp /= (float)(adjObjNum + 1);
        curr_tmp += HEAT_TRANSFER_SPEED * tex1Dfetch(texIn, idx);
        curr[idx] = fminf(curr_tmp, 1.0f);
    }
#else
    #if QUICK_TRANS
    __global__ void 
    propagate_Kernel(std::uint32_t num_objects, std::uint32_t* adjObjects, std::uint32_t* sortedIDs,
            std::uint32_t* prefix_sum, std::uint32_t* adjObjNums, float *prev, float* curr) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx > num_objects - 1) {
            return;
        }
        std::uint32_t adjObjNum = adjObjNums[idx];

        float curr_tmp = 0.6 * prev[idx]; // heat loss
        for (int i = 0; i < adjObjNum; i++) {
            curr_tmp += prev[adjObjects[prefix_sum[idx]/*offset*/ + i]];
        }
        curr_tmp /= (float)(adjObjNum + 1);
        curr_tmp += HEAT_TRANSFER_SPEED * prev[idx];
        curr[idx] = fminf(curr_tmp, 1.0f);
        
    }
    #else
    __global__ void 
    propagate_Kernel(std::uint32_t num_objects, std::uint32_t* adjObjects, std::uint32_t* sortedIDs,
            std::uint32_t* prefix_sum, std::uint32_t* adjObjNums, float *prev, float* curr) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx > num_objects - 1) {
            return;
        }
        std::uint32_t adjObjNum = adjObjNums[idx];

        curr[idx] = prev[idx];
        for (int i = 0; i < adjObjNum; i++) {
            curr[idx] += prev[adjObjects[prefix_sum[idx]/*offset*/ + i]];
        }
        curr[idx] /= (float)(adjObjNum + 1);
    }
    #endif
#endif




__host__ void
lbvh::BVH::propagate() {
    if (bvh_status != lbvh::BVH_STATUS::STATE_PROPAGATE) {
        printf("PROPAGATE Kernel needs to be called at SATAE_PROPAGATE.\n");
        return;
    }

    /* get prefix sum pointer */
    std::uint32_t* scan_res_ptr = thrust::raw_pointer_cast(scan_res_d_.data());
    std::uint32_t* adjObjNumList_raw_ptr = thrust::raw_pointer_cast(adjObjNumList_d_.data());

    /* kernel property */
    int threadsPerBlock = 256;
    int blocksPerGrid = (gNumObjects + threadsPerBlock - 1) / threadsPerBlock;
    #if STEAM_GPU
        HANDLE_ERROR(hipMemcpyAsync(gIntensityIn_d_, gIntensity_h_, 
                sizeof(float) * gNumObjects, hipMemcpyHostToDevice, stream1));
    #else
        HANDLE_ERROR(hipMemcpy(gIntensityIn_d_, gIntensity_h_, 
                sizeof(float) * gNumObjects, hipMemcpyHostToDevice));
    #endif
    #if TEXTURE_GPU
        propagate_Kernel<<<blocksPerGrid, threadsPerBlock>>>
                (gNumObjects, adjObjInfo_d_, gSortedObjIDs, scan_res_ptr, 
                    adjObjNumList_raw_ptr, gIntensityOut_d_);
    #else
        #if STEAM_GPU
        propagate_Kernel<<<blocksPerGrid, threadsPerBlock, 0, stream1>>>
                (gNumObjects, adjObjInfo_d_, gSortedObjIDs, scan_res_ptr, 
                    adjObjNumList_raw_ptr, gIntensityIn_d_, gIntensityOut_d_);
        #else
        propagate_Kernel<<<blocksPerGrid, threadsPerBlock>>>
                (gNumObjects, adjObjInfo_d_, gSortedObjIDs, scan_res_ptr, 
                    adjObjNumList_raw_ptr, gIntensityIn_d_, gIntensityOut_d_);
        #endif
    #endif
    HANDLE_ERROR(hipDeviceSynchronize());
    /* copy out calculated value */
    #if STEAM_GPU
    HANDLE_ERROR(hipMemcpyAsync(gIntensity_h_, gIntensityOut_d_,  
            gNumObjects * sizeof(float), hipMemcpyDeviceToHost, stream1));
    HANDLE_ERROR( hipStreamSynchronize( stream1 ) );
    #else
    HANDLE_ERROR(hipMemcpy(gIntensity_h_, gIntensityOut_d_,  
            gNumObjects * sizeof(float), hipMemcpyDeviceToHost));
    #endif
    
    for (int i = 0; i < HEAT_SOURCE_SIZE; i++) {
        gIntensity_h_[heatSource[i]] = 1.0;
    }

    /* swap in/out buffer */
    dstOut = 1 - dstOut;
}


void startHeatTransfer() {
    if (bvhInstance->getStatus() != lbvh::BVH_STATUS::STATE_PROPAGATE) {
        printf("Please complete prerequisites before propagating.\n");
        return;
    }
    if (dstOut == -1) {
        dstOut = 1;

        hipDeviceProp_t  prop;
        int whichDevice;
        HANDLE_ERROR( hipGetDevice( &whichDevice ) );
        HANDLE_ERROR( hipGetDeviceProperties( &prop, whichDevice ) );
        if (!prop.deviceOverlap) {
            printf( "Device will not handle overlaps, so no speed up from streams\n" );
            return;
        }

        /* allocate data */
        HANDLE_ERROR(hipHostAlloc((void**)&gIntensity_h_, gNumObjects * sizeof(float*), hipHostMallocDefault));
        HANDLE_ERROR(hipMalloc((void**)&gIntensityIn_d_, gNumObjects * sizeof(float)));
        HANDLE_ERROR(hipMalloc((void**)&gIntensityOut_d_, gNumObjects * sizeof(float)));

        #if STEAM_GPU
            HANDLE_ERROR( hipStreamCreate( &stream1 ) );
        #endif
        #if TEXTURE_GPU
            HANDLE_ERROR(hipBindTexture(NULL, texIn, gIntensityIn_d_, gNumObjects * sizeof(float)));
        #endif
        printf("--> Intensity memory has been allocated.\n");

        /* initialize the first iteration temperature */
        for (int i = 0; i < gNumObjects; i++) {
            gIntensity_h_[i] = 0;
        }

        for (int i = 0; i < HEAT_SOURCE_SIZE; i++) {
            gIntensity_h_[heatSource[i]] = 1.0;
        }
        // HANDLE_ERROR(hipMemcpy(gIntensityIn_d_, gIntensity_h_, sizeof(float) * gNumObjects, hipMemcpyHostToDevice));

        printf("--> Heat source IDs have been determined.\n");
        printf("--> Initialization done.\n");
        printf("--> propagating start...\n");
    }
    else {
        TIMING_BEGIN
        bvhInstance->propagate();
        TIMING_END("time cost:")
    }
}


void quit_heatTransfer() {

    HANDLE_ERROR(hipFree(gIntensity_h_));
    #if STEAM_GPU
        HANDLE_ERROR( hipStreamDestroy( stream1 ) );
    #endif
    #if TEXTURE_GPU
        HANDLE_ERROR(hipUnbindTexture(texIn));
    #endif
    HANDLE_ERROR(hipFree(gIntensityIn_d_));
    HANDLE_ERROR(hipFree(gIntensityOut_d_));
}
