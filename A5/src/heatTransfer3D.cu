#include "hip/hip_runtime.h"
#include "bvh.h"
#include "aabb.h"
#include "common.h"
#include <thrust/device_ptr.h>
#include <thrust/device_free.h>

#define HEAT_SOURCE_SIZE (3)
#define HEAT_TRANSFER_SPEED (0.04f)
#define QUICK_TRANS (0)

volatile int dstOut = -1;
float isource = 1.0f;

int heatSource[HEAT_SOURCE_SIZE] = {100, 10, 200};


/* heat and color */
float* gIntensity_h_;
float* gIntensityIn_d_;
float* gIntensityOut_d_;

/* global value */
extern std::uint32_t gNumObjects;
extern std::uint32_t* gSortedObjIDs;


lbvh::BVH* bvhInstance = lbvh::BVH::getInstance();
/* print info */
std::string div_signs(10, '-');

#if QUICK_TRANS
__global__ void 
propagate_Kernel(std::uint32_t num_objects, std::uint32_t* adjObjects, std::uint32_t* sortedIDs,
        std::uint32_t* prefix_sum, std::uint32_t* adjObjNums, float *prev, float* curr) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx > num_objects - 1) {
        return;
    }
    std::uint32_t adjObjNum = adjObjNums[idx];
    std::uint32_t sum = 0;
    
    for (int i = 0; i < adjObjNum; i++) {
        sum += prev[adjObjects[prefix_sum[idx]/*offset*/ + i]];
    }
    curr[idx] = prev[idx] + HEAT_TRANSFER_SPEED * (sum - adjObjNum * prev[idx]);
}
#else
__global__ void 
propagate_Kernel(std::uint32_t num_objects, std::uint32_t* adjObjects, std::uint32_t* sortedIDs,
        std::uint32_t* prefix_sum, std::uint32_t* adjObjNums, float *prev, float* curr) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx > num_objects - 1) {
        return;
    }
    std::uint32_t adjObjNum = adjObjNums[idx];

    curr[idx] = prev[idx];
    for (int i = 0; i < adjObjNum; i++) {
        curr[idx] += prev[adjObjects[prefix_sum[idx]/*offset*/ + i]];
    }
    curr[idx] /= (float)(adjObjNum + 1);
}
#endif

__host__ void
lbvh::BVH::propagate() {
    if (bvh_status != lbvh::BVH_STATUS::STATE_PROPAGATE) {
        printf("PROPAGATE Kernel needs to be called at SATAE_PROPAGATE.\n");
        return;
    }

    /* get prefix sum pointer */
    std::uint32_t* scan_res_ptr = thrust::raw_pointer_cast(scan_res_d_.data());
    std::uint32_t* adjObjNumList_raw_ptr = thrust::raw_pointer_cast(adjObjNumList_d_.data());

    // START_GPU
    /* kernel property */
    int threadsPerBlock = 256;
    int blocksPerGrid = (gNumObjects + threadsPerBlock - 1) / threadsPerBlock;
    if (dstOut == 1) {
        HANDLE_ERROR(hipMemcpy(gIntensityIn_d_, gIntensity_h_, sizeof(float) * gNumObjects, hipMemcpyHostToDevice));
        propagate_Kernel<<<blocksPerGrid, threadsPerBlock>>>
                (gNumObjects, adjObjInfo_d_, gSortedObjIDs, scan_res_ptr, adjObjNumList_raw_ptr, gIntensityIn_d_, gIntensityOut_d_);
    }
    else if (dstOut == 0) {
        HANDLE_ERROR(hipMemcpy(gIntensityOut_d_, gIntensity_h_, sizeof(float) * gNumObjects, hipMemcpyHostToDevice));
        propagate_Kernel<<<blocksPerGrid, threadsPerBlock>>>
                (gNumObjects, adjObjInfo_d_, gSortedObjIDs, scan_res_ptr, adjObjNumList_raw_ptr, gIntensityOut_d_, gIntensityIn_d_);
    }
    HANDLE_ERROR(hipDeviceSynchronize());

    /* copy out calculated value */
    if (dstOut == 1) {
        HANDLE_ERROR(hipMemcpy(gIntensity_h_, gIntensityOut_d_,  gNumObjects * sizeof(float), hipMemcpyDeviceToHost));
    } 
    else if (dstOut == 0) {
        HANDLE_ERROR(hipMemcpy(gIntensity_h_, gIntensityIn_d_,  gNumObjects * sizeof(float), hipMemcpyDeviceToHost));
    }
    // END_GPU
    for (int i = 0; i < HEAT_SOURCE_SIZE; i++) {
        gIntensity_h_[heatSource[i]] = 1.0;
    }

    /* swap in/out buffer */
    dstOut = 1 - dstOut;
}


void startHeatTransfer() {
    if (bvhInstance->getStatus() != lbvh::BVH_STATUS::STATE_PROPAGATE) {
        printf("Please complete prerequisites before propagating.\n");
        return;
    }
    if (dstOut == -1) {
        dstOut = 1;

        /* allocate data */
        gIntensity_h_ = (float*)malloc(gNumObjects * sizeof(float));
        HANDLE_ERROR(hipMalloc((void**)&gIntensityIn_d_, gNumObjects * sizeof(float)));
        HANDLE_ERROR(hipMalloc((void**)&gIntensityOut_d_, gNumObjects * sizeof(float)));
        printf("--> Intensity memory has been allocated.\n");

        /* initialize the first iteration temperature */
        for (int i = 0; i < gNumObjects; i++) {
            gIntensity_h_[i] = 0;
        }

        for (int i = 0; i < HEAT_SOURCE_SIZE; i++) {
            gIntensity_h_[heatSource[i]] = 1.0;
        }
        // HANDLE_ERROR(hipMemcpy(gIntensityIn_d_, gIntensity_h_, sizeof(float) * gNumObjects, hipMemcpyHostToDevice));

        printf("--> Heat source IDs have been determined.\n");
        printf("--> Initialization done.\n");
        printf("--> propagating start...\n");
    }
    else {
        TIMING_BEGIN
        bvhInstance->propagate();
        TIMING_END("time cost:")
    }
}


void quit_heatTransfer() {
    free(gIntensity_h_);
    HANDLE_ERROR(hipFree(gIntensityIn_d_));
    HANDLE_ERROR(hipFree(gIntensityOut_d_));
}
