#include "hip/hip_runtime.h"
#include "tiny_obj_loader.h"

#include "book.h"
#include "bvh.h"
#include "morton_code.h"

#include <hip/hip_runtime.h>
#include <>

#include <thrust/functional.h>
#include <thrust/for_each.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>

#include <sys/stat.h> // check file status

namespace lbvh {

/* get common upper bits */
__device__ __inline__ int 
commonUpperBits(const unsigned int lhs, const unsigned int rhs) noexcept {
    return __clz(lhs ^ rhs);
}

/* get cuda device info */
void cudaDevInfo();

/* Func declaration */
__device__ int2 
determineRange(std::uint32_t* sortedMortonCodes, int num_objects, int idx);

__device__ __inline__ int
findSplit(std::uint32_t* sortedMortonCodes, int first, int last);

/* Kernel declaration */
__global__ void 
computeBBoxes_kernel(const std::uint32_t num_objects, triangle_t* trianglePtr, vec3f* verticePtr, AABB* aabbPtr);

__global__ void 
computeMortonCode_kernel(std::uint32_t num_objects, std::uint32_t* objectIDs, 
                            AABB aabb_bound, AABB* aabbs, std::uint32_t* mortonCodes);

__global__ void
construtInternalNodes_kernel(std::uint32_t* sortedMortonCodes, std::uint32_t* sortedObjectIDs, int numObjects,
                            InternalNodePtr internalNodes, LeafNodePtr leafNodes, AABB* bboxes);

__global__ void
createAABBHierarchy_Kernel(int num_objects, LeafNodePtr leafNodes);

struct minUnaryFunc{
    __host__ __device__
    vec3f operator () (const AABB& a){
        return a.bmin;
    }
};

struct minBinaryFunc{
    __host__ __device__
    vec3f operator () (const vec3f& a, const vec3f& b){
        return vmin(a,b);
    }
};
struct maxUnaryFunc{
    
    __host__ __device__
    vec3f operator () (const AABB& a){
        return a.bmax;
    }
};

struct maxBinaryFunc{
    __host__ __device__
    vec3f operator () (const vec3f& a, const vec3f& b){
        return vmax(a,b);
    }
};


__host__ void 
BVH::loadObj(std::string& inputfile) {
    struct stat buffer;  
    if (stat(inputfile.c_str(), &buffer) != 0) {
        printf("No file found according to given argument <%s>.\n", inputfile.c_str());
        exit(EXIT_FAILURE);
    }


    std::string suffixStr = inputfile.substr(inputfile.find_last_of('.') + 1);
    if (suffixStr != std::string("obj")) {
        printf("Invalid file type. Please select .obj suffix file.\n");
        exit(EXIT_FAILURE);
    }

    /* stage info */
    std::string div_signs(10, '-');
    std::cout << div_signs << "  Stage 1: Loading objects  " << div_signs << std::endl;
    printf("Loading objcts from <%s> ...\n", inputfile.c_str());

    tinyobj::attrib_t attrib;
    std::vector<tinyobj::shape_t> shapes;
    std::vector<tinyobj::material_t> materials;
    
    std::string err;
    bool ret = tinyobj::LoadObj(&attrib, &shapes, &materials, &err, inputfile.c_str());

    /* deal with unexpected situation */
    if (!err.empty()) {
        std::cerr << err << std::endl;
    }

    if (!ret) {
        exit(EXIT_FAILURE);
    }

    // Loop over shapes
    size_t shapes_size = shapes.size();
    for (size_t s = 0; s < shapes_size; s++) {
        // Loop over faces (polygon)
        size_t index_offset = 0;
        size_t face_num = shapes[s].mesh.num_face_vertices.size(); // total face number
        for (size_t f = 0; f < face_num; f++) {
            /* store triangle's three vertices (a,b,c) index 
            ( index_t: vertex_index, normal_index, texcoord_index) */
            triangle_t tri;
            tri.a = shapes[s].mesh.indices[index_offset + 0];
            tri.b = shapes[s].mesh.indices[index_offset + 1];
            tri.c = shapes[s].mesh.indices[index_offset + 2];
            // triangle points property
            triangle_indices_h_.push_back(tri);
            // per-face material
            index_offset += 3;
        }
    }

    /* get vertices */
    size_t vertices_size = attrib.vertices.size();
    for (size_t s = 0; s < vertices_size; s += 3) {
        vec3f vertice;
        vertice.x = attrib.vertices.at(s + 0);
        vertice.y = attrib.vertices.at(s + 1);
        vertice.z = attrib.vertices.at(s + 2);
        vertices_h_.push_back(vertice);
    }

    /* get normals */
    size_t normals_size = attrib.normals.size();
    for (size_t s = 0; s < normals_size; s += 3) {
        vec3f normal;
        normal.x = attrib.normals.at(s + 0);
        normal.y = attrib.normals.at(s + 1);
        normal.z = attrib.normals.at(s + 2);
        normals_h_.push_back(normal);
    }
    
    printf("objcet size: %lu, vertices size: %lu, normals size: %lu.\n", 
                        triangle_indices_h_.size(), vertices_h_.size(), normals_h_.size());

    return;
}


__host__ void 
BVH::construct() {
    if(triangle_indices_h_.size() == 0u || 
        vertices_h_.size() == 0u || 
        normals_h_.size() == 0u ) {

        printf("Please load objects fisrt. Error happens in %s at line %d.\n", __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    std::string div_signs(10, '-');
    std::cout << div_signs << "  Start LBVH Construction  " << div_signs << std::endl;
    printf("[CUDA device information]\n");
    cudaDevInfo();

    /* basic information */
    const std::uint32_t num_objects        = triangle_indices_h_.size();
    const std::uint32_t num_internal_nodes = num_objects - 1;
    const std::uint32_t num_nodes          = num_objects * 2 - 1;
    /* kernel property */
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_objects - 1 + threadsPerBlock - 1) / threadsPerBlock;

    /* stage info assistant  */
    printf("[Kernel property]\n block:              %d\n threads:            %d\n\n", 
                blocksPerGrid, threadsPerBlock);
    printf("[LBVH property]\n leaf nodes:         %u\n internal nodes:     %u\n totol nodes:        %u\n", 
                num_objects, num_internal_nodes, num_nodes);
    printf(" num_objects size:   %lu bytes\n aabbs size:         %lu bytes\n vertices size:      %lu bytes\n\n", 
                num_objects * sizeof(triangle_t), num_objects * sizeof(AABB), normals_h_.size() * sizeof(vec3f));
    /* ---------------- STAGE 1: load objects ---------------- */
    /* allocte specific memory size */
    HANDLE_ERROR(hipMalloc((void**)&triangle_indices_d_, num_objects * sizeof(triangle_t)));
    HANDLE_ERROR(hipMalloc((void**)&vertices_d_, vertices_h_.size() * sizeof(vec3f)));
    HANDLE_ERROR(hipMalloc((void**)&normals_d_, normals_h_.size() * sizeof(vec3f)));
    HANDLE_ERROR(hipMalloc((void**)&aabbs, num_objects * sizeof(AABB)));


    /* copy data from host to device */
    HANDLE_ERROR(hipMemcpy(triangle_indices_d_, triangle_indices_h_.data(), 
                            num_objects * sizeof(triangle_t), 
                            hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMemcpy(vertices_d_, vertices_h_.data(), 
                            vertices_h_.size() * sizeof(vec3f), 
                            hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMemcpy(normals_d_, normals_h_.data(), 
                            normals_h_.size() * sizeof(vec3f), 
                            hipMemcpyHostToDevice));

    /* construct aabb */
    std::cout << div_signs << "  Stage 2: Compute AABB bounding boxes.  " << div_signs << std::endl;
    computeBBoxes_kernel<<<blocksPerGrid, threadsPerBlock>>>(num_objects, triangle_indices_d_, vertices_d_, aabbs);

    /* calculate morton code for all objects */
    AABB aabb_bound;
    thrust::device_ptr<AABB> aabb_d_ptr(aabbs);
    aabb_bound.bmax = thrust::transform_reduce(
        aabb_d_ptr, aabb_d_ptr + num_objects,
        maxUnaryFunc(),
        vec3f(-1e9f, -1e9f, -1e9f),
        maxBinaryFunc());
    aabb_bound.bmin = thrust::transform_reduce(
        aabb_d_ptr, aabb_d_ptr + num_objects,
        minUnaryFunc(),
        vec3f(1e9f, 1e9f, 1e9f),
        minBinaryFunc());

    printf("--> found AABB bound min(%0.6f, %0.6f , %0.6f)\n" , aabb_bound.bmin.x , aabb_bound.bmin.y , aabb_bound.bmin.z);
    printf("--> found AABB bound max(%0.6f, %0.6f , %0.6f)\n" , aabb_bound.bmax.x , aabb_bound.bmax.y , aabb_bound.bmax.z);

    /* ---------------- STAGE 2: build BVH Tree ---------------- */
    HANDLE_ERROR(hipMalloc(&mortonCodes, num_objects * sizeof(std::uint32_t)));
    HANDLE_ERROR(hipMalloc(&objectIDs, num_objects * sizeof(std::uint32_t)));
    HANDLE_ERROR(hipMalloc(&leafNodes, num_objects * sizeof(LeafNode)));
    HANDLE_ERROR(hipMalloc(&internalNodes, (num_objects - 1) * sizeof(InternalNode)));

    /* compute morton code */
    std::cout << div_signs << "  Stage 3: Calculate morton codes.  " << div_signs << std::endl;
    computeMortonCode_kernel<<<blocksPerGrid, threadsPerBlock>>>(num_objects, objectIDs, aabb_bound, aabbs, mortonCodes);

    /* sort morton codes */
    thrust::device_ptr<std::uint32_t> mortonCodes_d_ptr(mortonCodes);
    thrust::device_ptr<std::uint32_t> objectIDs_d_ptr(objectIDs);
    thrust::sort_by_key(mortonCodes_d_ptr, mortonCodes_d_ptr + num_objects, objectIDs_d_ptr);
    printf("--> morton codes have been sorted.\n");

    std::cout << div_signs << "  Stage 4: Construct LBVH hierarchy.  " << div_signs << std::endl;
    /* construct leaf nodes */
    thrust::device_ptr<LeafNode> leafNodes_d_ptr(leafNodes);
    thrust::transform(objectIDs_d_ptr, objectIDs_d_ptr + num_objects, leafNodes_d_ptr,
        [] __device__ (const std::uint32_t idx) {
            LeafNode leaf;
            leaf.objectID = idx;
            
            return leaf;
        });
    printf("--> leaf nodes have been constructed.\n");
    /* construct internal nodes */
    construtInternalNodes_kernel<<<blocksPerGrid, threadsPerBlock>>>(mortonCodes, objectIDs, num_objects, internalNodes, leafNodes, aabbs);
    printf("--> internal nodes have been constructed.\n");

    /* create AABB for each node by bottom-up strategy */
    createAABBHierarchy_Kernel<<<blocksPerGrid, threadsPerBlock>>>(num_objects, leafNodes);
    printf("--> lbvh hierarchy has been constructed.\n");

    return;
}


BVH::~BVH() {
    HANDLE_ERROR(hipFree(triangle_indices_d_));
    HANDLE_ERROR(hipFree(vertices_d_));
    HANDLE_ERROR(hipFree(normals_d_));
    HANDLE_ERROR(hipFree(aabbs));
}

/**
 * @brief In order to construct a binary radix tree, we need to determine 
 * the range of keys covered by each internal node, as well as its children. 
 * 
 * @param sortedMortonCodes morton codes which have been sorted
 * @param num_objects leaf node number
 * @param idx thread or object ID
 * @return range 
 */
__device__ int2 
determineRange(std::uint32_t* sortedMortonCodes, int num_objects, int idx) {
    // determine the range of keys covered by each internal node (as well as its children)
    // direction is found by looking at the neighboring keys ki-1 , ki , ki+1
    // the index is either the beginning of the range or the end of the range
    if (idx == 0) {
        return make_int2(0, num_objects - 1);
    } // When Idx = 0, this means the range cover the whole array

    int direction = 0;
    int commonPrefix_L = 0;
    int commonPrefix_R = 0;

    /* get current key_idx neighbors' common prefixes and then determine the direction
    so that we can get the minimum common prefix according to direction */
    commonPrefix_R = commonUpperBits(sortedMortonCodes[idx], sortedMortonCodes[idx+1]);
    commonPrefix_L = commonUpperBits(sortedMortonCodes[idx], sortedMortonCodes[idx-1]);
    direction = commonPrefix_L - commonPrefix_R > 0 ? -1 : 1;

    int commonPrefix_min = commonUpperBits(sortedMortonCodes[idx], sortedMortonCodes[idx - direction]);

    /* find the upper bound roughly, exponentially increasing step until the condition is violiated. */
    int lmax = 2;
    int largerCommonPrefixDir_index = idx + lmax * direction;
    while ( largerCommonPrefixDir_index >= 0 && 
            largerCommonPrefixDir_index < num_objects && 
            commonUpperBits(sortedMortonCodes[idx], sortedMortonCodes[largerCommonPrefixDir_index]) > commonPrefix_min) {
        lmax *= 2;
        largerCommonPrefixDir_index = idx + lmax * direction;
    }

    /* find the other end using binary search, this will get a preciser bound */
    int l = 0;
    int t = lmax >> 1;
    while (t > 0) {
        largerCommonPrefixDir_index = idx + (l + t) * direction;
        if (largerCommonPrefixDir_index >= 0 &&
            largerCommonPrefixDir_index < num_objects &&
            commonUpperBits(sortedMortonCodes[idx], sortedMortonCodes[largerCommonPrefixDir_index]) > commonPrefix_min) {
                l = l + t;
        }
        t = t >> 1;
    }
    /* precise upper bound index */
    int jdx = idx + l * direction;

    /* make sure that idx < jdx */
    if (direction < 0) {
        thrust::swap(idx, jdx);
    }

    return make_int2(idx, jdx);
}



__device__ __inline__ int
findSplit(std::uint32_t* sortedMortonCodes, int first, int last) {
    // Identical Morton codes => split the range in the middle.
    unsigned int firstCode = sortedMortonCodes[first];
    unsigned int lastCode = sortedMortonCodes[last];

    if (firstCode == lastCode) {
        return (first + last) >> 1;
    }

    // Calculate the number of highest bits that are the same
    // for all objects, using the count-leading-zeros intrinsic.
    int commonPrefix = commonUpperBits(firstCode, lastCode);

    // Use binary search to find where the next bit differs.
    // Specifically, we are looking for the highest object that
    // shares more than commonPrefix bits with the first one.
    int split = first; // initial guess
    int step = last - first;

    do {
        step = (step + 1) >> 1; // exponential decrease
        int newSplit = split + step; // proposed new position

        if (newSplit < last) {
            std::uint32_t splitCode = sortedMortonCodes[newSplit];
            int splitPrefix = commonUpperBits(firstCode, splitCode);
            if (splitPrefix > commonPrefix) {
                split = newSplit; // accept proposal
            }
        }
    }
    while (step > 1);

    return split;
}


__global__ void 
computeBBoxes_kernel(const std::uint32_t num_objects, triangle_t* triangles, vec3f* vertices, AABB* aabbs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx > num_objects - 1) {
        return;
    } // leaf node index range [0, n - 1]

    aabbs[idx].bmax = vmax(vertices[triangles[idx].a.vertex_index], vertices[triangles[idx].b.vertex_index]);
    aabbs[idx].bmin = vmin(vertices[triangles[idx].a.vertex_index], vertices[triangles[idx].b.vertex_index]);
    aabbs[idx].bmax = vmax(vertices[triangles[idx].c.vertex_index], aabbs[idx].bmax);
    aabbs[idx].bmin = vmin(vertices[triangles[idx].c.vertex_index], aabbs[idx].bmin);
}



__global__ void 
computeMortonCode_kernel(std::uint32_t num_objects, std::uint32_t* objectIDs, 
                            AABB aabb_bound, AABB* aabbs, std::uint32_t* mortonCodes) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx > num_objects - 1) {
        return; 
    } // leaf node index range [0, n - 1]

    objectIDs[idx] = idx;
    vec3f centroid = aabbs[idx].getCentroid();
    centroid.x = (centroid.x - aabb_bound.bmin.x) / (aabb_bound.bmax.x - aabb_bound.bmin.x);
    centroid.y = (centroid.y - aabb_bound.bmin.y) / (aabb_bound.bmax.y - aabb_bound.bmin.y);
    centroid.z = (centroid.z - aabb_bound.bmin.z) / (aabb_bound.bmax.z - aabb_bound.bmin.z);
    mortonCodes[idx] = morton3D(centroid);

    return;
}

__global__ void
construtInternalNodes_kernel(std::uint32_t* sortedMortonCodes, std::uint32_t* sortedObjectIDs, int numObjects,
                            InternalNodePtr internalNodes, LeafNodePtr leafNodes, AABB* bboxes) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx > numObjects - 2) {
        return;
    } // internal nodes index range [0, n - 2]

    /* Find out which range of objects the node corresponds to. */
    int2 range = determineRange(sortedMortonCodes, numObjects, idx);
    int first = range.x;
    int last = range.y;

    /* Determine where to split the range. */
    int split = findSplit(sortedMortonCodes, first, last);

    // Select childA.
    NodePtr childA;
    if (split == first) {
        childA = &leafNodes[split];
    } // only one node remained, so that this node must be a leaf node
    else {
        childA = &internalNodes[split];
    } 

    // Select childB.
    NodePtr childB;
    if (split + 1 == last) {
        childB = &leafNodes[split + 1];
    }
    else {
        childB = &internalNodes[split + 1];
    }

    // Record parent-child relationships.
    internalNodes[idx].childA = childA;
    internalNodes[idx].childB = childB;
    childA->parent = &internalNodes[idx];
    childB->parent = &internalNodes[idx];
    
    // Node 0 is the root.
}


__global__ void
createAABBHierarchy_Kernel(int num_objects, LeafNodePtr leafNodes) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx > num_objects - 1)
        return;

    Node* nodeIdxParent = (leafNodes + idx)->parent;

    while (nodeIdxParent != nullptr) {
        const int old = atomicCAS(&nodeIdxParent->updateFlag, 0, 1);
        if (old == 0) {
            /* first thread entered here. 
                Wait the other thread from the other child node. */ 
            return;
        }
        assert(old == 1);
        /* old has been one, another thead can access here. merge its child's AABB boxes. */
        nodeIdxParent->bbox = merge(nodeIdxParent->childA->bbox, nodeIdxParent->childB->bbox);
        /* reading global memory is a blocking process, but writing action doesn't. The thread
            will continue working rather than wait until the writing completed. */
        __threadfence();
        /* get next parent */
        nodeIdxParent = nodeIdxParent->parent;
    }

    return;
}

void cudaDevInfo() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    int dev;
    for (dev = 0; dev < deviceCount; dev++)
    {
        int driver_version(0), runtime_version(0);

        size_t available, total;
        hipMemGetInfo(&available, &total);

        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        if (dev == 0)
            if (deviceProp.minor = 9999 && deviceProp.major == 9999)
                printf("\n");

        printf("\nDevice%d:\"%s\"\n", dev, deviceProp.name);
        hipDriverGetVersion(&driver_version);
        printf("CUDA Driver Version:                            %d.%d\n", 
            driver_version / 1000, (driver_version % 1000) / 10);
        hipRuntimeGetVersion(&runtime_version);
        printf("CUDA Runtime Version:                           %d.%d\n", 
            runtime_version / 1000, (runtime_version % 1000) / 10);
        printf("Device Prop:                                    %d.%d\n", 
            deviceProp.major, deviceProp.minor);
        printf("Total amount of Global Memory:                  %lu bytes\n", 
            deviceProp.totalGlobalMem);
        printf("Total amount of AVALUABLE Memory:               %lu bytes\n", 
            available);
        printf("Number of SMs:                                  %d\n", 
            deviceProp.multiProcessorCount);
        printf("Total amount of Constant Memory:                %lu bytes\n", 
            deviceProp.totalConstMem);
        printf("Total amount of Shared Memory per block:        %lu bytes\n", 
            deviceProp.sharedMemPerBlock);
        printf("Total number of registers available per block:  %d\n", 
            deviceProp.regsPerBlock);
        printf("Warp size:                                      %d\n", 
            deviceProp.warpSize);
        printf("Maximum number of threads per SM:               %d\n", 
            deviceProp.maxThreadsPerMultiProcessor);
        printf("Maximum number of threads per block:            %d\n", 
            deviceProp.maxThreadsPerBlock);
        printf("Maximum size of each dimension of a block:      %d x %d x %d\n", 
            deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
        printf("Maximum size of each dimension of a grid:       %d x %d x %d\n", 
            deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
        printf("Maximum memory pitch:                           %lu bytes\n", 
            deviceProp.memPitch);
        printf("Texture alignmemt:                              %lu bytes\n", 
            deviceProp.texturePitchAlignment);
        printf("Clock rate:                                     %.2f GHz\n", 
            deviceProp.clockRate * 1e-6f);
        printf("Memory Clock rate:                              %.0f MHz\n", 
            deviceProp.memoryClockRate * 1e-3f);
        printf("Memory Bus Width:                               %d-bit\n\n", 
            deviceProp.memoryBusWidth);
    }
}

}





